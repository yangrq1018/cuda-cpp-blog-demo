#include "hip/hip_runtime.h"
#include <iostream>
#include "timer.h"

// Kernel function to add the elements of two arrays
__global__ void _add(int n, float *x, float *y)
{
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    int stride = blockDim.x * gridDim.x;
    for (int i = index; i < n; i += stride)
        y[i] = x[i] + y[i];
}

void add(int n, float *x, float *y)
{
    float *temp_x, *temp_y;
    hipMalloc(&temp_x, n * sizeof(float));
    hipMalloc(&temp_y, n * sizeof(float));

    // copy input to device memory
    hipMemcpy(temp_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(temp_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    // Run kernel on 1M elements on the GPU
    int blockSize = 256;
    int numBlocks = (n + blockSize - 1) / blockSize; // round up in case N is not a multiple of blockSize

    {
        Timer timer("kernel");
        _add<<<numBlocks, blockSize>>>(n, temp_x, temp_y);
    }

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // copy result back
    hipMemcpy(y, temp_y, n * sizeof(float), hipMemcpyDeviceToHost);

    // Free memory
    hipFree(temp_x);
    hipFree(temp_y);
}